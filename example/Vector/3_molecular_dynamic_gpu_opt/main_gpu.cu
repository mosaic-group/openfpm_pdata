
#include <hip/hip_runtime.h>
/*!
 * \page Vector_3_md_dyn_gpu_opt Vector 3 molecular dynamic on GPU (optimized version)
 *
 * [TOC]
 *
 * # Molecular Dynamic with Lennard-Jones potential on GPU (Optimized) {#e3_md_gpu_opt}
 *
 * \htmlonly
 * <img src="http://openfpm.mpi-cbg.de/web/images/examples/3_md_gpu/md_4_GPU.png"/>
 * \endhtmlonly
 *
 * In this optimized version of the example \ref e3_md_gpu, we operate two optimization:
 *
 * * We make the access coalesced
 * * We use half radius cell-list spacing
 *
 * ## Coalesced access {#e3_gpu_opt_ca}
 *
 * In GPU to get the maximum performance it is very important to access in a coalesced way. Access in a coalesced way mean that
 *  if thread 1 access adress 0x000 thread 2 (in the same Streaming Multiprocessors) should ideally access 0x004 or more in
 *  general an adress in the same cache line. Another factor that contribute to speed is to overall restrict the threads in
 *  the same SM should to possibly work on a limited number of caches lines so that the L1 cache of each SM could optimally
 *  speed up the access to global memory.
 *
 *  Unfortunately particles by nature can be randomly distributed in space and memory, and reach the ideal situation
 * in the case of neighborhood access of the particles can be challenging. Suppose that thread 1 take particle 0 and thread 2
 * take particle 1, but 0 and 1 are far in space the neighborhood of 0 does not overlap the neighborhood of 1. This mean that
 * most probably the access of the neighborhood of 0 will be scattered in memory and the same for 1, having an extremely low
 * probability that 2 thread in the SM hit the same cache line and increasing the number of cache lines the SM has to retrieve.
 *  On the other hand if we reorder the particle in memory by their spatial position using a cell list like in figure.
 *
 * \htmlonly
  <table style="width:100%">
  <tr>
    <td><img src="http://openfpm.mpi-cbg.de/web/images/examples/3_md_gpu/vector_sorted.jpg"/></td>
    <td><img src="http://openfpm.mpi-cbg.de/web/images/examples/3_md_gpu/vector_unsorted.jpg"/></td>
  </tr>
  <tr>
    <td><strong>Fig1: Sorted vector</strong></td>
    <td><strong>Fig2: Unsorted vector</strong></td>
  </tr>
</table>
  \endhtmlonly
 *
 * We can see that now the neighborhood of particle 0 and particle 1 overlap increasing that chance of cache hit, additionally if
 * all particles processed by one SM stay in one cell or few neighborhood cell, the number of cache line that an SM has to read is
 * reduced, with a significant speed-up.
 *
 * In OpenFPM get a Cell-list produce a re-ordered version of the original vector by default. It is possible to offload the sorted
 *  version vector_dist_gpu instead of the normal one using the function \b toKernel_sorted() \b instead of the function \b toKernel \b.
 *
 * \snippet Vector/3_molecular_dynamic_gpu_opt/main_gpu.cu calc_force_sorted
 *
 * The rest remain mainly the same, with the expectation, that we now use the macro GET_PARTICLE_SORT. This macro is similar to GET_PARTICLE
 * but with a substantial difference. While in the normal unsorted vector particles in the ghost area are always added at the end
 * in the sorted one domain + ghost are reordered, and there is not a clear separation between them. This mean that we need a list of all
 * the domain particles, if we want iterate cross them. GET_PARTICLE_SORT use a list to convert thread index to domain particle index.
 * Additionally when we get a neighborhood iterator from the Cell-list we must use \bget_sorted_index\b instead of \bget\b
 *
 * \snippet Vector/3_molecular_dynamic_gpu_opt/main_gpu.cu get_sorted_index
 *
 * After we launched the kernel all the data are written in the sorted vector. In order to merge back the data to the unsorted one
 * we have to use the function \b vd.merge_sort<force>(NN) \b. Where vd is the vector_dist_gpu where we want to merge the
 * data from sorted to non sorted. \b force \b is the property we want to merge and \b NN \b is the Cell-list that produced the
 * sorted distribution.
 *
 * \snippet Vector/3_molecular_dynamic_gpu_opt/main_gpu.cu merge_sort
 *
 * \note it is possible to launch multiple kernel on the sorted version, but consider that at some point the data must be merged
 * back because functions like map and ghost_get work on the unsorted version
 *
 * ## Half radius cell-list spacing {#e3_gpu_opt_hr}
 *
 * Using Cell-lists with spacing equal to the radius in general require to fetch all the 9 cells in 2D and 27 cells in 3D. All the
 * particles in such cells include particles within radius r and others more distant than r. This mean that we have to filter the particles
 * checking the radius. It is possible to filter further more the particles using finer cell-list cells. Suppose that you use
 * cell-lists with spacing half of the radius. we just the to check the 25 cells in 2D and the 125 cells in 3D. While we have more
 * cells the overall volume spanned by the 25/125 cells is just a fraction. In fact the surface of the 25 cells is given by
 *
 * \f$ (5\frac{h}{2})^2 = \frac{25}{4} h^2 \f$
 * \f$ (5\frac{h}{2})^3 = \frac{125}{8} h^3 \f$
 *
 * while for the normal cell-list is
 *
 * \f$ (3h)^2 = 9h^2 \f$
 * \f$ (3h)^3 = 27h^3 \f$
 *
 * This mean that the finer cell-list in order to find the neighborhood particles use an area smaller: precisely is 69% of
 * the normal cell-list in 2D, and 57% of the normal cell-list in 3D. In particles this mean that normal cell-list return
 * in average 45% more particles in 2D and  75% more in 3D.
 *
 * Constructing an half spacing cell-list is standard. In the function \b getCellListGPU \b we specify half radius
 *
 * \snippet Vector/3_molecular_dynamic_gpu_opt/main_gpu.cu get_half_cl
 *
 * while to use it, instead of the \b getNNIteratorBox \b we use
 *
 * \note \b getNNIteratorBox \b is initialized in cell-list by \b setBoxNN \b (default = 1) before it transferred to gpu that indicate how many neighborhood cell the NN iterator has to span.
 *       For example \b setBoxNN(1) \b is the standard 9/27 neighborhood cell-list.\b setBoxNN(2) \b is the 25/125 neighborhood
 *        and so on.
 *
 */

#ifdef __NVCC__

#include "Vector/vector_dist.hpp"
#include "Plot/GoogleChart.hpp"
#include "Plot/util.hpp"
#include "timer.hpp"

#ifdef TEST_RUN
size_t nstep = 100;
#else
size_t nstep = 1000;
#endif

typedef float real_number;

constexpr int velocity = 0;
constexpr int force = 1;
constexpr int energy = 2;

template<typename vector_dist_type,typename NN_type>
__global__ void calc_force_gpu(vector_dist_type vd, NN_type NN, real_number sigma12, real_number sigma6, real_number r_cut2)
{
	unsigned int p;
	GET_PARTICLE_SORT(p,NN);

	// Get the position xp of the particle
	Point<3,real_number> xp = vd.getPos(p);

	// Reset the force counter
	vd.template getProp<force>(p)[0] = 0.0;
	vd.template getProp<force>(p)[1] = 0.0;
	vd.template getProp<force>(p)[2] = 0.0;

	Point<3,real_number> force_;
	force_.get(0) = 0.0;
	force_.get(1) = 0.0;
	force_.get(2) = 0.0;

	// Get an iterator over the neighborhood particles of p
	auto Np = NN.getNNIteratorBox(NN.getCell(vd.getPos(p)));

	// For each neighborhood particle ...
	while (Np.isNext())
	{
		//! \cond [get_sorted_index] \endcond

		// ... q
		auto q = Np.get_sort();

		//! \cond [get_sorted_index] \endcond

		// if (p == q) skip this particle
		if (q == p)	{++Np; continue;};

		// Get the position of p
		Point<3,real_number> xq = vd.getPos(q);

		// Get the distance between p and q
		Point<3,real_number> r = xp - xq;

		// take the norm of this vector
		real_number rn = norm2(r);

		if (rn > r_cut2)
		{++Np; continue;};

		// Calculate the force, using pow is slower
		Point<3,real_number> f = 24.0*(2.0 *sigma12 / (rn*rn*rn*rn*rn*rn*rn) -  sigma6 / (rn*rn*rn*rn)) * r;
		force_ += f;

		// Next neighborhood
		++Np;
	}

	// we sum the force produced by q on p
	vd.template getProp<force>(p)[0] = force_.get(0);
	vd.template getProp<force>(p)[1] = force_.get(1);
	vd.template getProp<force>(p)[2] = force_.get(2);
}

template<typename vector_dist_type>
__global__ void update_velocity_position(vector_dist_type vd, real_number dt)
{
	auto p = GET_PARTICLE(vd);

	// here we calculate v(tn + 0.5)
	vd.template getProp<velocity>(p)[0] += 0.5*dt*vd.template getProp<force>(p)[0];
	vd.template getProp<velocity>(p)[1] += 0.5*dt*vd.template getProp<force>(p)[1];
	vd.template getProp<velocity>(p)[2] += 0.5*dt*vd.template getProp<force>(p)[2];

	// here we calculate x(tn + 1)
	vd.getPos(p)[0] += vd.template getProp<velocity>(p)[0]*dt;
	vd.getPos(p)[1] += vd.template getProp<velocity>(p)[1]*dt;
	vd.getPos(p)[2] += vd.template getProp<velocity>(p)[2]*dt;
}

template<typename vector_dist_type>
__global__ void update_velocity(vector_dist_type vd, real_number dt)
{
	auto p = GET_PARTICLE(vd);

	// here we calculate v(tn + 1)
	vd.template getProp<velocity>(p)[0] += 0.5*dt*vd.template getProp<force>(p)[0];
	vd.template getProp<velocity>(p)[1] += 0.5*dt*vd.template getProp<force>(p)[1];
	vd.template getProp<velocity>(p)[2] += 0.5*dt*vd.template getProp<force>(p)[2];
}

template<typename vector_dist_type,typename NN_type>
__global__ void particle_energy(vector_dist_type vd, NN_type NN, real_number sigma12, real_number sigma6, real_number shift, real_number r_cut2)
{
	unsigned int p;
	GET_PARTICLE_SORT(p,NN);

	// Get the position of the particle p
	Point<3,real_number> xp = vd.getPos(p);

	// Get an iterator over the neighborhood of the particle p
	auto Np = NN.getNNIteratorBox(NN.getCell(vd.getPos(p)));

	real_number E = 0;

	// For each neighborhood of the particle p
	while (Np.isNext())
	{
		// Neighborhood particle q
		auto q = Np.get_sort();

		// if p == q skip this particle
		if (q == p)	{++Np; continue;};

		// Get position of the particle q
		Point<3,real_number> xq = vd.getPos(q);

		// take the normalized direction
		real_number rn = norm2(xp - xq);

		if (rn > r_cut2)
		{++Np;continue;}

		// potential energy (using pow is slower)
		E += 2.0 * ( sigma12 / (rn*rn*rn*rn*rn*rn) - sigma6 / ( rn*rn*rn) ) - shift;

		// Next neighborhood
		++Np;
	}

	// Kinetic energy of the particle given by its actual speed
	vd.template getProp<energy>(p) = E + (vd.template getProp<velocity>(p)[0]*vd.template getProp<velocity>(p)[0] +
			vd.template getProp<velocity>(p)[1]*vd.template getProp<velocity>(p)[1] +
			vd.template getProp<velocity>(p)[2]*vd.template getProp<velocity>(p)[2]) / 2;
}

template<typename CellList> void calc_forces(vector_dist_gpu<3,real_number, aggregate<real_number[3],real_number[3],real_number> > & vd, CellList & NN, real_number sigma12, real_number sigma6, real_number r_cut2)
{
	vd.updateCellList(NN);

	// Get an iterator over particles
	auto it2 = vd.getDomainIteratorGPU();

	//! \cond [calc_force_sorted] \endcond

	CUDA_LAUNCH(calc_force_gpu,it2,vd.toKernel_sorted(),NN.toKernel(),sigma12,sigma6,r_cut2);

	//! \cond [calc_force_sorted] \endcond

	//! \cond [merge_sort] \endcond

	vd.merge_sort<force>(NN);

	//! \cond [merge_sort] \endcond
}

template<typename CellList> real_number calc_energy(vector_dist_gpu<3,real_number, aggregate<real_number[3],real_number[3],real_number> > & vd, CellList & NN, real_number sigma12, real_number sigma6, real_number r_cut2)
{
	real_number rc = r_cut2;
	real_number shift = 2.0 * ( sigma12 / (rc*rc*rc*rc*rc*rc) - sigma6 / ( rc*rc*rc) );

	vd.updateCellList(NN);

	auto it2 = vd.getDomainIteratorGPU();

	CUDA_LAUNCH(particle_energy,it2,vd.toKernel_sorted(),NN.toKernel(),sigma12,sigma6,shift,r_cut2);

	vd.merge_sort<energy>(NN);

	// Calculated energy
	return reduce_local<energy,_add_>(vd);
}

int main(int argc, char* argv[])
{
	openfpm_init(&argc,&argv);

	real_number sigma = 0.01;
	real_number r_cut =3.0*sigma;

	// we will use it do place particles on a 10x10x10 Grid like
	size_t sz[3] = {100,100,100};

	// domain
	Box<3,float> box({0.0,0.0,0.0},{1.0,1.0,1.0});

	// Boundary conditions
	size_t bc[3]={PERIODIC,PERIODIC,PERIODIC};

	// ghost, big enough to contain the interaction radius
	Ghost<3,float> ghost(r_cut);

	real_number dt = 0.00005;
	real_number sigma12 = pow(sigma,12);
	real_number sigma6 = pow(sigma,6);

	openfpm::vector<real_number> x;
	openfpm::vector<openfpm::vector<real_number>> y;

	vector_dist_gpu<3,real_number, aggregate<real_number[3],real_number[3],real_number> > vd(0,box,bc,ghost);

	// We create the grid iterator
	auto it = vd.getGridIterator(sz);

	while (it.isNext())
	{
		// Create a new particle
		vd.add();

		// key contain (i,j,k) index of the grid
		auto key = it.get();

		// The index of the grid can be accessed with key.get(0) == i, key.get(1) == j ...
		// We use getLastPos to set the position of the last particle added
		vd.getLastPos()[0] = key.get(0) * it.getSpacing(0);
		vd.getLastPos()[1] = key.get(1) * it.getSpacing(1);
		vd.getLastPos()[2] = key.get(2) * it.getSpacing(2);

		// We use getLastProp to set the property value of the last particle we added
		vd.template getLastProp<velocity>()[0] = 0.0;
		vd.template getLastProp<velocity>()[1] = 0.0;
		vd.template getLastProp<velocity>()[2] = 0.0;

		vd.template getLastProp<force>()[0] = 0.0;
		vd.template getLastProp<force>()[1] = 0.0;
		vd.template getLastProp<force>()[2] = 0.0;

		++it;
	}

	vd.hostToDevicePos();
	vd.hostToDeviceProp<velocity,force>();

	vd.map(RUN_ON_DEVICE);
	vd.ghost_get<>(RUN_ON_DEVICE);

	timer tsim;
	tsim.start();

	//! \cond [md steps] \endcond

	//! \cond [get_half_cl] \endcond

	// Get the Cell list structure
	auto NN = vd.getCellListGPU(r_cut / 2.0);

	//! \cond [get_half_cl] \endcond

	// The standard
	// auto NN = vd.getCellList(r_cut);

	// calculate forces
	calc_forces(vd,NN,sigma12,sigma6,r_cut*r_cut);
	unsigned long int f = 0;

	// MD time stepping
	for (size_t i = 0; i < nstep ; i++)
	{
		// Get the iterator
		auto it3 = vd.getDomainIteratorGPU();

		CUDA_LAUNCH(update_velocity_position,it3,vd.toKernel(),dt);

		// Because we moved the particles in space we have to map them and re-sync the ghost
		vd.map(RUN_ON_DEVICE);
		vd.template ghost_get<>(RUN_ON_DEVICE);

		// calculate forces or a(tn + 1) Step 2
		calc_forces(vd,NN,sigma12,sigma6,r_cut*r_cut);

		// Integrate the velocity Step 3
		auto it4 = vd.getDomainIteratorGPU();

		CUDA_LAUNCH(update_velocity,it4,vd.toKernel(),dt);

		// After every iteration collect some statistic about the configuration
		if (i % 1000 == 0)
		{
			vd.deviceToHostPos();
			vd.deviceToHostProp<0,1,2>();

			// We write the particle position for visualization (Without ghost)
			vd.deleteGhost();
			vd.write_frame("particles_",f);

			// we resync the ghost
			vd.ghost_get<>(RUN_ON_DEVICE);

			// We calculate the energy
			real_number energy = calc_energy(vd,NN,sigma12,sigma6,r_cut*r_cut);
			auto & vcl = create_vcluster();
			vcl.sum(energy);
			vcl.execute();

			// we save the energy calculated at time step i c contain the time-step y contain the energy
			x.add(i);
			y.add({energy});

			// We also print on terminal the value of the energy
			// only one processor (master) write on terminal
			if (vcl.getProcessUnitID() == 0)
				std::cout << "Energy: " << energy << std::endl;

			f++;
		}
	}

	tsim.stop();
	std::cout << "Time: " << tsim.getwct() << std::endl;

	// Google charts options, it store the options to draw the X Y graph
	GCoptions options;

	// Title of the graph
	options.title = std::string("Energy with time");

	// Y axis name
	options.yAxis = std::string("Energy");

	// X axis name
	options.xAxis = std::string("iteration");

	// width of the line
	options.lineWidth = 1.0;

	// Resolution in x
	options.width = 1280;

	// Resolution in y
	options.heigh = 720;

	// Add zoom capability
	options.more = GC_ZOOM;

	// Object that draw the X Y graph
	GoogleChart cg;

	// Add the graph
	// The graph that it produce is in svg format that can be opened on browser
	cg.AddLinesGraph(x,y,options);

	// Write into html format
	cg.write("gc_plot2_out.html");

	openfpm_finalize();
}

#else

int main(int argc, char* argv[])
{
        return 0;
}

#endif


