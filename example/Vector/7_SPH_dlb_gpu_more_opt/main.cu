
#include <hip/hip_runtime.h>
/*! \page Vector_7_sph_dlb_gpu_more_opt Vector 7 SPH Dam break simulation with Dynamic load balacing on Multi-GPU (more optimized version)
 *
 *
 * [TOC]
 *
 *
 * # SPH with Dynamic load Balancing on GPU (More Optimized) # {#SPH_dlb_gpu_more_opt}
 *
 *
 * This example show the classical SPH Dam break simulation with load balancing and dynamic load balancing. The main difference with
 * \ref{SPH_dlb_gpu_opt} is that here we use 2 kernel to calculate forces one for fluid and one for boundaries. Also we use the function
 * get_indexes_by_type to get the indexes of the fluid and boundary particles and use these two set to launch two distinct kernel
 * (one over fluid and one over boundary) to calculate forces and density change. set. Simulate 1.5 second should be duable on mobile
 *  1050Ti in about 1 hour and 7 minutes
 *
 * \htmlonly
 * <a href="#" onclick="hide_show('vector-video-3')" >Simulation video 1</a><br>
 * <div style="display:none" id="vector-video-3">
 * <video id="vid3" width="1200" height="576" controls> <source src="http://openfpm.mpi-cbg.de/web/images/examples/7_SPH_dlb/sph_gpu1.mp4" type="video/mp4"></video>
 * </div>
 * <a href="#" onclick="hide_show('vector-video-4')" >Simulation video 2</a><br>
 * <div style="display:none" id="vector-video-4">
 * <video id="vid4" width="1200" height="576" controls> <source src="http://openfpm.mpi-cbg.de/web/images/examples/7_SPH_dlb/sph_gpu2.mp4" type="video/mp4"></video>
 * </div>
 * <a href="#" onclick="hide_show('vector-video-15')" >Simulation video 3</a><br>
 * <div style="display:none" id="vector-video-15">
 * <video id="vid15" width="1200" height="576" controls> <source src="http://openfpm.mpi-cbg.de/web/images/examples/7_SPH_dlb/sph_gpu3.mp4" type="video/mp4"></video>
 * </div>
 * \endhtmlonly
 *
 *
 * ## get_indexes_by_type ## {#e7_sph_more_opt_gibt}
 *
 * This function can be used to get the indexes of a certain type on a particle set and save such indexes in an openfpm::vector<aggregate<unsigned int>>
 * the constructed set of indices can be used to run a kernel on a specific set of particles.
 *
 * \snippet Vector/7_SPH_dlb_gpu_more_opt/main.cu get indexes by type
 *
 * the function get_indexes_by_type has three arguments the first is the vector of the properties of the particles. In
 * this case because we use the sorted particles to calculate forces, so we have to get the indexes for the sorted
 * particles with vd.getPropVectorSort(). In case we want to use the non sorted we use vd.getPropVector(). The second
 * argument is the output containing the indexes of the particles types we want to get. Because the vector can contain
 * ghost particles and real particles setting with the third argument we indicate we want only real particles and no ghost particles
 * The last argument is the GPU context handle
 *
 * we report the full code here
 *
 *
 */

#ifdef __NVCC__

#define PRINT_STACKTRACE
#define STOP_ON_ERROR
#define OPENMPI
#define SCAN_WITH_CUB
#define SORT_WITH_CUB
//#define SE_CLASS1

//#define USE_LOW_REGISTER_ITERATOR

#include "Vector/vector_dist.hpp"
#include <math.h>
#include "Draw/DrawParticles.hpp"



typedef float real_number;

// A constant to indicate boundary particles
#define BOUNDARY 0

// A constant to indicate fluid particles
#define FLUID 1

// initial spacing between particles dp in the formulas
const real_number dp = 0.00425;
// Maximum height of the fluid water
// is going to be calculated and filled later on
real_number h_swl = 0.0;

// c_s in the formulas (constant used to calculate the sound speed)
const real_number coeff_sound = 20.0;

// gamma in the formulas
const real_number gamma_ = 7.0;

// sqrt(3.0*dp*dp) support of the kernel
const real_number H = 0.00736121593217;

// Eta in the formulas
const real_number Eta2 = 0.01 * H*H;

const real_number FourH2 = 4.0 * H*H;

// alpha in the formula
const real_number visco = 0.1;

// cbar in the formula (calculated later)
real_number cbar = 0.0;

// Mass of the fluid particles
const real_number MassFluid = 0.0000767656;

// Mass of the boundary particles
const real_number MassBound = 0.0000767656;

//

// End simulation time
#ifdef TEST_RUN
const real_number t_end = 0.001;
#else
const real_number t_end = 1.5;
#endif

// Gravity acceleration
const real_number gravity = 9.81;

// Reference densitu 1000Kg/m^3
const real_number rho_zero = 1000.0;

// Filled later require h_swl, it is b in the formulas
real_number B = 0.0;

// Constant used to define time integration
const real_number CFLnumber = 0.2;

// Minimum T
const real_number DtMin = 0.00001;

// Minimum Rho allowed
const real_number RhoMin = 700.0;

// Maximum Rho allowed
const real_number RhoMax = 1300.0;

// Filled in initialization
real_number max_fluid_height = 0.0;

// Properties

// FLUID or BOUNDARY
const size_t type = 0;

// Density
const int rho = 1;

// Density at step n-1
const int rho_prev = 2;

// Pressure
const int Pressure = 3;

// Delta rho calculated in the force calculation
const int drho = 4;

// calculated force
const int force = 5;

// velocity
const int velocity = 6;

// velocity at previous step
const int velocity_prev = 7;

const int red = 8;

const int red2 = 9;

// Type of the vector containing particles
typedef vector_dist_gpu<3,real_number,aggregate<unsigned int,real_number,  real_number,    real_number,     real_number,     real_number[3], real_number[3], real_number[3], real_number, real_number>> particles;
//                                              |          |             |               |                |                |               |               |               |            |
//                                              |          |             |               |                |                |               |               |               |            |
//                                             type      density       density        Pressure          delta            force          velocity        velocity        reduction     another
//                                                                     at n-1                           density                                         at n - 1        buffer        reduction buffer


struct ModelCustom
{
	template<typename Decomposition, typename vector> inline void addComputation(Decomposition & dec,
			                                                                     vector & vd,
																				 size_t v,
																				 size_t p)
	{
		if (vd.template getProp<type>(p) == FLUID)
			dec.addComputationCost(v,4);
		else
			dec.addComputationCost(v,3);
	}

	template<typename Decomposition> inline void applyModel(Decomposition & dec, size_t v)
	{
		dec.setSubSubDomainComputationCost(v, dec.getSubSubDomainComputationCost(v) * dec.getSubSubDomainComputationCost(v));
	}

	real_number distributionTol()
	{
		return 1.01;
	}
};

template<typename vd_type>
__global__ void EqState_gpu(vd_type vd, real_number B)
{
	auto a = GET_PARTICLE(vd);

	real_number rho_a = vd.template getProp<rho>(a);
	real_number rho_frac = rho_a / rho_zero;

	vd.template getProp<Pressure>(a) = B*( rho_frac*rho_frac*rho_frac*rho_frac*rho_frac*rho_frac*rho_frac - 1.0);
}

inline void EqState(particles & vd)
{
	auto it = vd.getDomainIteratorGPU();

	CUDA_LAUNCH(EqState_gpu,it,vd.toKernel(),B);
}


const real_number a2 = 1.0/M_PI/H/H/H;

inline __device__ __host__ real_number Wab(real_number r)
{
	r /= H;

	if (r < 1.0)
		return (1.0 - 3.0/2.0*r*r + 3.0/4.0*r*r*r)*a2;
	else if (r < 2.0)
		return (1.0/4.0*(2.0 - r*r)*(2.0 - r*r)*(2.0 - r*r))*a2;
	else
		return 0.0;
}


const real_number c1 = -3.0/M_PI/H/H/H/H;
const real_number d1 = 9.0/4.0/M_PI/H/H/H/H;
const real_number c2 = -3.0/4.0/M_PI/H/H/H/H;
const real_number a2_4 = 0.25*a2;
// Filled later
real_number W_dap = 0.0;

inline __device__ __host__ void DWab(Point<3,real_number> & dx, Point<3,real_number> & DW, real_number r)
{
	const real_number qq=r/H;

    real_number qq2 = qq * qq;
    real_number fac1 = (c1*qq + d1*qq2)/r;
    real_number b1 = (qq < 1.0f)?1.0f:0.0f;

    real_number wqq = (2.0f - qq);
    real_number fac2 = c2 * wqq * wqq / r;
    real_number b2 = (qq >= 1.0f && qq < 2.0f)?1.0f:0.0f;

    real_number factor = (b1*fac1 + b2*fac2);

    DW.get(0) = factor * dx.get(0);
    DW.get(1) = factor * dx.get(1);
    DW.get(2) = factor * dx.get(2);
}

// Tensile correction
inline __device__ __host__  real_number Tensile(real_number r, real_number rhoa, real_number rhob, real_number prs1, real_number prs2, real_number W_dap)
{
	const real_number qq=r/H;
	//-Cubic Spline kernel
	real_number wab;
	if(r>H)
	{
		real_number wqq1=2.0f-qq;
		real_number wqq2=wqq1*wqq1;

		wab=a2_4*(wqq2*wqq1);
	}
	else
	{
	    real_number wqq2=qq*qq;
	    real_number wqq3=wqq2*qq;

	    wab=a2*(1.0f-1.5f*wqq2+0.75f*wqq3);
	}

	//-Tensile correction.
	real_number fab=wab*W_dap;
	fab*=fab; fab*=fab; //fab=fab^4
	const real_number tensilp1=(prs1/(rhoa*rhoa))*(prs1>0.0f? 0.01f: -0.2f);
	const real_number tensilp2=(prs2/(rhob*rhob))*(prs2>0.0f? 0.01f: -0.2f);

	return (fab*(tensilp1+tensilp2));
}


inline __device__ __host__ real_number Pi(const Point<3,real_number> & dr, real_number rr2, Point<3,real_number> & dv, real_number rhoa, real_number rhob, real_number massb, real_number cbar, real_number & visc)
{
	const real_number dot = dr.get(0)*dv.get(0) + dr.get(1)*dv.get(1) + dr.get(2)*dv.get(2);
	const real_number dot_rr2 = dot/(rr2+Eta2);
	visc=(dot_rr2 < visc)?visc:dot_rr2;

	if(dot < 0)
	{
		const float amubar=H*dot_rr2;
		const float robar=(rhoa+rhob)*0.5f;
		const float pi_visc=(-visco*cbar*amubar/robar);

		return pi_visc;
    }
	else
		return 0.0f;
}

template<typename particles_type, typename fluid_ids_type,typename NN_type>
__global__ void calc_forces_fluid_gpu(particles_type vd, fluid_ids_type fids, NN_type NN, real_number W_dap, real_number cbar)
{
	// ... a
	unsigned int a;

	GET_PARTICLE_BY_ID(a,fids);

	real_number max_visc = 0.0f;

	// Get the position xp of the particle
	Point<3,real_number> xa = vd.getPos(a);

	// Type of the particle
	unsigned int typea = vd.template getProp<type>(a);

	// Get the density of the of the particle a
	real_number rhoa = vd.template getProp<rho>(a);

	// Get the pressure of the particle a
	real_number Pa = vd.template getProp<Pressure>(a);

	// Get the Velocity of the particle a
	Point<3,real_number> va = vd.template getProp<velocity>(a);

	Point<3,real_number> force_;
	force_.get(0) = 0.0f;
	force_.get(1) = 0.0f;
	force_.get(2) = -gravity;
	real_number drho_ = 0.0f;

	// Get an iterator over the neighborhood particles of p
	auto Np = NN.getNNIteratorBox(NN.getCell(xa));

	// For each neighborhood particle
	while (Np.isNext() == true)
	{
		// ... q
		auto b = Np.get_sort();

		// Get the position xp of the particle
		Point<3,real_number> xb = vd.getPos(b);

		// if (p == q) skip this particle this condition should be done in the r^2 = 0
		//if (a == b)	{++Np; continue;};

        unsigned int typeb = vd.template getProp<type>(b);

        real_number massb = (typeb == FLUID)?MassFluid:MassBound;
        Point<3,real_number> vb = vd.template getProp<velocity>(b);
        real_number Pb = vd.template getProp<Pressure>(b);
        real_number rhob = vd.template getProp<rho>(b);

		// Get the distance between p and q
		Point<3,real_number> dr = xa - xb;
		Point<3,real_number> v_rel = va - vb;
		// take the norm of this vector
		real_number r2 = norm2(dr);

		// if they interact
		if (r2 < FourH2 && r2 >= 1e-16)
		{
			real_number r = sqrtf(r2);

			Point<3,real_number> DW;
			DWab(dr,DW,r);

			real_number factor = - massb*((Pa + Pb) / (rhoa * rhob) + Tensile(r,rhoa,rhob,Pa,Pb,W_dap) + Pi(dr,r2,v_rel,rhoa,rhob,massb,cbar,max_visc));

			// Bound - Bound does not produce any change
			factor = (typea == BOUNDARY && typeb == BOUNDARY)?0.0f:factor;

			force_.get(0) += factor * DW.get(0);
			force_.get(1) += factor * DW.get(1);
			force_.get(2) += factor * DW.get(2);

			real_number scal = massb*(v_rel.get(0)*DW.get(0)+v_rel.get(1)*DW.get(1)+v_rel.get(2)*DW.get(2));
			scal = (typea == BOUNDARY && typeb == BOUNDARY)?0.0f:scal;

			drho_ += scal;
		}

		++Np;
	}

	vd.template getProp<red>(a) = max_visc;

	vd.template getProp<force>(a)[0] = force_.get(0);
	vd.template getProp<force>(a)[1] = force_.get(1);
	vd.template getProp<force>(a)[2] = force_.get(2);
	vd.template getProp<drho>(a) = drho_;
}

template<typename particles_type, typename fluid_ids_type,typename NN_type>
__global__ void calc_forces_border_gpu(particles_type vd, fluid_ids_type fbord, NN_type NN, real_number W_dap, real_number cbar)
{
	// ... a
	unsigned int a;

	GET_PARTICLE_BY_ID(a,fbord);

	real_number max_visc = 0.0f;

	// Get the position xp of the particle
	Point<3,real_number> xa = vd.getPos(a);

	// Type of the particle
	unsigned int typea = vd.template getProp<type>(a);

	// Get the Velocity of the particle a
	Point<3,real_number> va = vd.template getProp<velocity>(a);

	real_number drho_ = 0.0f;

	// Get an iterator over the neighborhood particles of p
	auto Np = NN.getNNIteratorBox(NN.getCell(xa));

	// For each neighborhood particle
	while (Np.isNext() == true)
	{
		// ... q
		auto b = Np.get_sort();

		// Get the position xp of the particle
		Point<3,real_number> xb = vd.getPos(b);

		// if (p == q) skip this particle this condition should be done in the r^2 = 0
		//if (a == b)	{++Np; continue;};

        unsigned int typeb = vd.template getProp<type>(b);

        real_number massb = (typeb == FLUID)?MassFluid:MassBound;
        Point<3,real_number> vb = vd.template getProp<velocity>(b);

		// Get the distance between p and q
		Point<3,real_number> dr = xa - xb;
		Point<3,real_number> v_rel = va - vb;
		// take the norm of this vector
		real_number r2 = norm2(dr);

		// if they interact
		if (r2 < FourH2 && r2 >= 1e-16)
		{
			real_number r = sqrtf(r2);

			Point<3,real_number> DW;
			DWab(dr,DW,r);

			real_number scal = massb*(v_rel.get(0)*DW.get(0)+v_rel.get(1)*DW.get(1)+v_rel.get(2)*DW.get(2));
			scal = (typea == BOUNDARY && typeb == BOUNDARY)?0.0f:scal;

			drho_ += scal;
		}

		++Np;
	}

	vd.template getProp<red>(a) = max_visc;

	vd.template getProp<drho>(a) = drho_;
}

struct type_is_fluid
{
	__device__ static bool check(int c)
	{
		return c == FLUID;
	}
};

struct type_is_border
{
        __device__ static bool check(int c)
        {
                return c == BOUNDARY;
        }
};

template<typename CellList> inline void calc_forces(particles & vd, CellList & NN, real_number & max_visc, size_t cnt, openfpm::vector_gpu<aggregate<int>> & fluid_ids, openfpm::vector_gpu<aggregate<int>> & border_ids)
{
	// Update the cell-list
	vd.updateCellList<type,rho,Pressure,velocity>(NN);

	//! \cond [get indexes by type] \endcond

	// get the particles fluid ids
	get_indexes_by_type<type,type_is_fluid>(vd.getPropVectorSort(),fluid_ids,vd.size_local(),vd.getVC().getGpuContext());

	// get the particles fluid ids
	get_indexes_by_type<type,type_is_border>(vd.getPropVectorSort(),border_ids,vd.size_local(),vd.getVC().getGpuContext());

	auto part = fluid_ids.getGPUIterator(96);
	CUDA_LAUNCH(calc_forces_fluid_gpu,part,vd.toKernel_sorted(),fluid_ids.toKernel(),NN.toKernel(),W_dap,cbar);

	part = border_ids.getGPUIterator(96);
	CUDA_LAUNCH(calc_forces_border_gpu,part,vd.toKernel_sorted(),border_ids.toKernel(),NN.toKernel(),W_dap,cbar);

	//! \cond [get indexes by type] \endcond

	vd.merge_sort<force,drho,red>(NN);

	max_visc = reduce_local<red,_max_>(vd);
}

template<typename vector_type>
__global__ void max_acceleration_and_velocity_gpu(vector_type vd)
{
	auto a = GET_PARTICLE(vd);

	Point<3,real_number> acc(vd.template getProp<force>(a));
	vd.template getProp<red>(a) = norm(acc);

	Point<3,real_number> vel(vd.template getProp<velocity>(a));
	vd.template getProp<red2>(a) = norm(vel);
}

void max_acceleration_and_velocity(particles & vd, real_number & max_acc, real_number & max_vel)
{
	// Calculate the maximum acceleration
	auto part = vd.getDomainIteratorGPU();

	CUDA_LAUNCH(max_acceleration_and_velocity_gpu,part,vd.toKernel());

	max_acc = reduce_local<red,_max_>(vd);
	max_vel = reduce_local<red2,_max_>(vd);

	Vcluster<> & v_cl = create_vcluster();
	v_cl.max(max_acc);
	v_cl.max(max_vel);
	v_cl.execute();
}


real_number calc_deltaT(particles & vd, real_number ViscDtMax)
{
	real_number Maxacc = 0.0;
	real_number Maxvel = 0.0;
	max_acceleration_and_velocity(vd,Maxacc,Maxvel);

	//-dt1 depends on force per unit mass.
	const real_number dt_f = (Maxacc)?sqrt(H/Maxacc):std::numeric_limits<float>::max();

	//-dt2 combines the Courant and the viscous time-step controls.
	const real_number dt_cv = H/(std::max(cbar,Maxvel*10.f) + H*ViscDtMax);

	//-dt new value of time step.
	real_number dt=real_number(CFLnumber)*std::min(dt_f,dt_cv);
	if(dt<real_number(DtMin))
	{dt=real_number(DtMin);}

	return dt;
}

template<typename vector_dist_type>
__global__ void verlet_int_gpu(vector_dist_type vd, real_number dt, real_number dt2, real_number dt205)
{
	// ... a
	auto a = GET_PARTICLE(vd);

	// if the particle is boundary
	if (vd.template getProp<type>(a) == BOUNDARY)
	{
		// Update rho
		real_number rhop = vd.template getProp<rho>(a);

		// Update only the density
    	vd.template getProp<velocity>(a)[0] = 0.0;
    	vd.template getProp<velocity>(a)[1] = 0.0;
    	vd.template getProp<velocity>(a)[2] = 0.0;
    	real_number rhonew = vd.template getProp<rho_prev>(a) + dt2*vd.template getProp<drho>(a);
    	vd.template getProp<rho>(a) = (rhonew < rho_zero)?rho_zero:rhonew;

	    vd.template getProp<rho_prev>(a) = rhop;

	    vd.template getProp<red>(a) = 0;

		return;
	}

	//-Calculate displacement and update position
	real_number dx = vd.template getProp<velocity>(a)[0]*dt + vd.template getProp<force>(a)[0]*dt205;
    real_number dy = vd.template getProp<velocity>(a)[1]*dt + vd.template getProp<force>(a)[1]*dt205;
    real_number dz = vd.template getProp<velocity>(a)[2]*dt + vd.template getProp<force>(a)[2]*dt205;

    vd.getPos(a)[0] += dx;
    vd.getPos(a)[1] += dy;
    vd.getPos(a)[2] += dz;

    real_number velX = vd.template getProp<velocity>(a)[0];
    real_number velY = vd.template getProp<velocity>(a)[1];
    real_number velZ = vd.template getProp<velocity>(a)[2];

    real_number rhop = vd.template getProp<rho>(a);

	vd.template getProp<velocity>(a)[0] = vd.template getProp<velocity_prev>(a)[0] + vd.template getProp<force>(a)[0]*dt2;
	vd.template getProp<velocity>(a)[1] = vd.template getProp<velocity_prev>(a)[1] + vd.template getProp<force>(a)[1]*dt2;
	vd.template getProp<velocity>(a)[2] = vd.template getProp<velocity_prev>(a)[2] + vd.template getProp<force>(a)[2]*dt2;
	vd.template getProp<rho>(a) = vd.template getProp<rho_prev>(a) + dt2*vd.template getProp<drho>(a);

    // Check if the particle go out of range in space and in density
    if (vd.getPos(a)[0] <  0.0 || vd.getPos(a)[1] < 0.0 || vd.getPos(a)[2] < 0.0 ||
        vd.getPos(a)[0] >  1.61 || vd.getPos(a)[1] > 0.68 || vd.getPos(a)[2] > 0.50 ||
		vd.template getProp<rho>(a) < RhoMin || vd.template getProp<rho>(a) > RhoMax)
    {
    	vd.template getProp<red>(a) = 1;
    }
    else
    {
    	vd.template getProp<red>(a) = 0;
    }


    vd.template getProp<velocity_prev>(a)[0] = velX;
    vd.template getProp<velocity_prev>(a)[1] = velY;
    vd.template getProp<velocity_prev>(a)[2] = velZ;
    vd.template getProp<rho_prev>(a) = rhop;
}

size_t cnt = 0;

void verlet_int(particles & vd, real_number dt)
{
	// particle iterator
	auto part = vd.getDomainIteratorGPU();

	real_number dt205 = dt*dt*0.5;
	real_number dt2 = dt*2.0;

	CUDA_LAUNCH(verlet_int_gpu,part,vd.toKernel(),dt,dt2,dt205);

	// remove the particles marked
	remove_marked<red>(vd);

	// increment the iteration counter
	cnt++;
}

template<typename vector_type>
__global__ void euler_int_gpu(vector_type vd,real_number dt, real_number dt205)
{
	// ... a
	auto a = GET_PARTICLE(vd);

	// if the particle is boundary
	if (vd.template getProp<type>(a) == BOUNDARY)
	{
		// Update rho
		real_number rhop = vd.template getProp<rho>(a);

		// Update only the density
    	vd.template getProp<velocity>(a)[0] = 0.0;
    	vd.template getProp<velocity>(a)[1] = 0.0;
    	vd.template getProp<velocity>(a)[2] = 0.0;
    	real_number rhonew = vd.template getProp<rho>(a) + dt*vd.template getProp<drho>(a);
    	vd.template getProp<rho>(a) = (rhonew < rho_zero)?rho_zero:rhonew;

	    vd.template getProp<rho_prev>(a) = rhop;

	    vd.template getProp<red>(a) = 0;

		return;
	}

	//-Calculate displacement and update position / Calcula desplazamiento y actualiza posicion.
	real_number dx = vd.template getProp<velocity>(a)[0]*dt + vd.template getProp<force>(a)[0]*dt205;
    real_number dy = vd.template getProp<velocity>(a)[1]*dt + vd.template getProp<force>(a)[1]*dt205;
    real_number dz = vd.template getProp<velocity>(a)[2]*dt + vd.template getProp<force>(a)[2]*dt205;

    vd.getPos(a)[0] += dx;
    vd.getPos(a)[1] += dy;
    vd.getPos(a)[2] += dz;

    real_number velX = vd.template getProp<velocity>(a)[0];
    real_number velY = vd.template getProp<velocity>(a)[1];
    real_number velZ = vd.template getProp<velocity>(a)[2];
    real_number rhop = vd.template getProp<rho>(a);

	vd.template getProp<velocity>(a)[0] = vd.template getProp<velocity>(a)[0] + vd.template getProp<force>(a)[0]*dt;
	vd.template getProp<velocity>(a)[1] = vd.template getProp<velocity>(a)[1] + vd.template getProp<force>(a)[1]*dt;
   	vd.template getProp<velocity>(a)[2] = vd.template getProp<velocity>(a)[2] + vd.template getProp<force>(a)[2]*dt;
   	vd.template getProp<rho>(a) = vd.template getProp<rho>(a) + dt*vd.template getProp<drho>(a);

    // Check if the particle go out of range in space and in density
    if (vd.getPos(a)[0] <  0.0 || vd.getPos(a)[1] < 0.0 || vd.getPos(a)[2] < 0.0 ||
        vd.getPos(a)[0] >  1.61 || vd.getPos(a)[1] > 0.68 || vd.getPos(a)[2] > 0.50 ||
		vd.template getProp<rho>(a) < RhoMin || vd.template getProp<rho>(a) > RhoMax)
    {vd.template getProp<red>(a) = 1;}
    else
    {vd.template getProp<red>(a) = 0;}

    vd.template getProp<velocity_prev>(a)[0] = velX;
    vd.template getProp<velocity_prev>(a)[1] = velY;
    vd.template getProp<velocity_prev>(a)[2] = velZ;
    vd.template getProp<rho_prev>(a) = rhop;
}

void euler_int(particles & vd, real_number dt)
{

	// particle iterator
	auto part = vd.getDomainIteratorGPU();

	real_number dt205 = dt*dt*0.5;

	CUDA_LAUNCH(euler_int_gpu,part,vd.toKernel(),dt,dt205);

	// remove the particles
	remove_marked<red>(vd);

	cnt++;
}

template<typename vector_type, typename NN_type>
__global__ void sensor_pressure_gpu(vector_type vd, NN_type NN, Point<3,real_number> probe, real_number * press_tmp)
{
	real_number tot_ker = 0.0;

	// Get the position of the probe i
	Point<3,real_number> xp = probe;

	// get the iterator over the neighbohood particles of the probes position
	auto itg = NN.getNNIteratorBox(NN.getCell(xp));
	while (itg.isNext())
	{
		auto q = itg.get_sort();

		// Only the fluid particles are importants
		if (vd.template getProp<type>(q) != FLUID)
		{
			++itg;
			continue;
		}

		// Get the position of the neighborhood particle q
		Point<3,real_number> xq = vd.getPos(q);

		// Calculate the contribution of the particle to the pressure
		// of the probe
		real_number r = sqrt(norm2(xp - xq));

		real_number ker = Wab(r) * (MassFluid / rho_zero);

		// Also keep track of the calculation of the summed
		// kernel
		tot_ker += ker;

		// Add the total pressure contribution
		*press_tmp += vd.template getProp<Pressure>(q) * ker;

		// next neighborhood particle
		++itg;
	}

	// We calculate the pressure normalizing the
	// sum over all kernels
	if (tot_ker == 0.0)
	{*press_tmp = 0.0;}
	else
	{*press_tmp = 1.0 / tot_ker * *press_tmp;}
}

template<typename Vector, typename CellList>
inline void sensor_pressure(Vector & vd,
                            CellList & NN,
                            openfpm::vector<openfpm::vector<real_number>> & press_t,
                            openfpm::vector<Point<3,real_number>> & probes)
{
    Vcluster<> & v_cl = create_vcluster();

    press_t.add();

    for (size_t i = 0 ; i < probes.size() ; i++)
    {
    	// A float variable to calculate the pressure of the problem
    	CudaMemory press_tmp_(sizeof(real_number));
    	real_number press_tmp;

        // if the probe is inside the processor domain
		if (vd.getDecomposition().isLocal(probes.get(i)) == true)
		{
			CUDA_LAUNCH_DIM3(sensor_pressure_gpu,1,1,vd.toKernel_sorted(),NN.toKernel(),probes.get(i),(real_number *)press_tmp_.toKernel());

			vd.merge<Pressure>(NN);

			// move calculated pressure on
			press_tmp_.deviceToHost();
			press_tmp = *(real_number *)press_tmp_.getPointer();
		}

		// This is not necessary in principle, but if you
		// want to make all processor aware of the history of the calculated
		// pressure we have to execute this
		v_cl.sum(press_tmp);
		v_cl.execute();

		// We add the calculated pressure into the history
		press_t.last().add(press_tmp);
	}
}

int main(int argc, char* argv[])
{
    // initialize the library
	openfpm_init(&argc,&argv);

	openfpm::vector_gpu<aggregate<int>> fluid_ids;
	openfpm::vector_gpu<aggregate<int>> border_ids;

#ifdef CUDIFY_USE_CUDA
	cudaDeviceSetCacheConfig(cudaFuncCachePreferL1);
#endif

	// It contain for each time-step the value detected by the probes
	openfpm::vector<openfpm::vector<real_number>> press_t;
	openfpm::vector<Point<3,real_number>> probes;

	probes.add({0.8779,0.3,0.02});
	probes.add({0.754,0.31,0.02});

	// Here we define our domain a 2D box with internals from 0 to 1.0 for x and y
	Box<3,real_number> domain({-0.05,-0.05,-0.05},{1.7010,0.7065,0.511});
	size_t sz[3] = {413,179,133};

	// Fill W_dap
	W_dap = 1.0/Wab(H/1.5);

	// Here we define the boundary conditions of our problem
    size_t bc[3]={NON_PERIODIC,NON_PERIODIC,NON_PERIODIC};

	// extended boundary around the domain, and the processor domain
	Ghost<3,real_number> g(2*H);

	particles vd(0,domain,bc,g,DEC_GRAN(128));

	//! \cond [draw fluid] \endcond

	// You can ignore all these dp/2.0 is a trick to reach the same initialization
	// of Dual-SPH that use a different criteria to draw particles
	Box<3,real_number> fluid_box({dp/2.0,dp/2.0,dp/2.0},{0.4+dp/2.0,0.67-dp/2.0,0.3+dp/2.0});

	// return an iterator to the fluid particles to add to vd
	auto fluid_it = DrawParticles::DrawBox(vd,sz,domain,fluid_box);

	// here we fill some of the constants needed by the simulation
	max_fluid_height = fluid_it.getBoxMargins().getHigh(2);
	h_swl = fluid_it.getBoxMargins().getHigh(2) - fluid_it.getBoxMargins().getLow(2);
	B = (coeff_sound)*(coeff_sound)*gravity*h_swl*rho_zero / gamma_;
	cbar = coeff_sound * sqrt(gravity * h_swl);

	// for each particle inside the fluid box ...
	while (fluid_it.isNext())
	{
		// ... add a particle ...
		vd.add();

		// ... and set it position ...
		vd.getLastPos()[0] = fluid_it.get().get(0);
		vd.getLastPos()[1] = fluid_it.get().get(1);
		vd.getLastPos()[2] = fluid_it.get().get(2);

		// and its type.
		vd.template getLastProp<type>() = FLUID;

		// We also initialize the density of the particle and the hydro-static pressure given by
		//
		// rho_zero*g*h = P
		//
		// rho_p = (P/B + 1)^(1/Gamma) * rho_zero
		//

		vd.template getLastProp<Pressure>() = rho_zero * gravity *  (max_fluid_height - fluid_it.get().get(2));

		vd.template getLastProp<rho>() = pow(vd.template getLastProp<Pressure>() / B + 1, 1.0/gamma_) * rho_zero;
		vd.template getLastProp<rho_prev>() = vd.template getLastProp<rho>();
		vd.template getLastProp<velocity>()[0] = 0.0;
		vd.template getLastProp<velocity>()[1] = 0.0;
		vd.template getLastProp<velocity>()[2] = 0.0;

		vd.template getLastProp<velocity_prev>()[0] = 0.0;
		vd.template getLastProp<velocity_prev>()[1] = 0.0;
		vd.template getLastProp<velocity_prev>()[2] = 0.0;

		// next fluid particle
		++fluid_it;
	}

	// Recipient
	Box<3,real_number> recipient1({0.0,0.0,0.0},{1.6+dp/2.0,0.67+dp/2.0,0.4+dp/2.0});
	Box<3,real_number> recipient2({dp,dp,dp},{1.6-dp/2.0,0.67-dp/2.0,0.4+dp/2.0});

	Box<3,real_number> obstacle1({0.9,0.24-dp/2.0,0.0},{1.02+dp/2.0,0.36,0.45+dp/2.0});
	Box<3,real_number> obstacle2({0.9+dp,0.24+dp/2.0,0.0},{1.02-dp/2.0,0.36-dp,0.45-dp/2.0});
	Box<3,real_number> obstacle3({0.9+dp,0.24,0.0},{1.02,0.36,0.45});

	openfpm::vector<Box<3,real_number>> holes;
	holes.add(recipient2);
	holes.add(obstacle1);
	auto bound_box = DrawParticles::DrawSkin(vd,sz,domain,holes,recipient1);

	while (bound_box.isNext())
	{
		vd.add();

		vd.getLastPos()[0] = bound_box.get().get(0);
		vd.getLastPos()[1] = bound_box.get().get(1);
		vd.getLastPos()[2] = bound_box.get().get(2);

		vd.template getLastProp<type>() = BOUNDARY;
		vd.template getLastProp<rho>() = rho_zero;
		vd.template getLastProp<rho_prev>() = rho_zero;
		vd.template getLastProp<velocity>()[0] = 0.0;
		vd.template getLastProp<velocity>()[1] = 0.0;
		vd.template getLastProp<velocity>()[2] = 0.0;

		vd.template getLastProp<velocity_prev>()[0] = 0.0;
		vd.template getLastProp<velocity_prev>()[1] = 0.0;
		vd.template getLastProp<velocity_prev>()[2] = 0.0;

		++bound_box;
	}

	auto obstacle_box = DrawParticles::DrawSkin(vd,sz,domain,obstacle2,obstacle1);

	while (obstacle_box.isNext())
	{
		vd.add();

		vd.getLastPos()[0] = obstacle_box.get().get(0);
		vd.getLastPos()[1] = obstacle_box.get().get(1);
		vd.getLastPos()[2] = obstacle_box.get().get(2);

		vd.template getLastProp<type>() = BOUNDARY;
		vd.template getLastProp<rho>() = rho_zero;
		vd.template getLastProp<rho_prev>() = rho_zero;
		vd.template getLastProp<velocity>()[0] = 0.0;
		vd.template getLastProp<velocity>()[1] = 0.0;
		vd.template getLastProp<velocity>()[2] = 0.0;

		vd.template getLastProp<velocity_prev>()[0] = 0.0;
		vd.template getLastProp<velocity_prev>()[1] = 0.0;
		vd.template getLastProp<velocity_prev>()[2] = 0.0;

		++obstacle_box;
	}

	vd.map();

	// Now that we fill the vector with particles
	ModelCustom md;

	vd.addComputationCosts(md);
	vd.getDecomposition().decompose();
	vd.map();

	///////////////////////////

	// Ok the initialization is done on CPU on GPU we are doing the main loop, so first we offload all properties on GPU

	vd.hostToDevicePos();
	vd.template hostToDeviceProp<type,rho,rho_prev,Pressure,velocity>();


	vd.ghost_get<type,rho,Pressure,velocity>(RUN_ON_DEVICE);

	auto NN = vd.getCellListGPU/*<CELLLIST_GPU_SPARSE<3,float>>*/(2*H / 2.0);
	NN.setBoxNN(2);

	timer tot_sim;
	tot_sim.start();

	size_t write = 0;
	size_t it = 0;
	size_t it_reb = 0;
	real_number t = 0.0;
	while (t <= t_end)
	{
		Vcluster<> & v_cl = create_vcluster();
		timer it_time;
		it_time.start();

		////// Do rebalancing every 200 timesteps
		it_reb++;
		if (it_reb == 300)
		{
			vd.map(RUN_ON_DEVICE);

			// Rebalancer for now work on CPU , so move to CPU
            vd.deviceToHostPos();
            vd.template deviceToHostProp<type>();

			it_reb = 0;
			ModelCustom md;
			vd.addComputationCosts(md);
			vd.getDecomposition().decompose();

			if (v_cl.getProcessUnitID() == 0)
			{std::cout << "REBALANCED " << it_reb << std::endl;}
		}

		vd.map(RUN_ON_DEVICE);

		// Calculate pressure from the density
		EqState(vd);

		real_number max_visc = 0.0;

		vd.ghost_get<type,rho,Pressure,velocity>(RUN_ON_DEVICE);


		// Calc forces
		calc_forces(vd,NN,max_visc,cnt,fluid_ids,border_ids);

		// Get the maximum viscosity term across processors
		v_cl.max(max_visc);
		v_cl.execute();

		// Calculate delta t integration
		real_number dt = calc_deltaT(vd,max_visc);

		// VerletStep or euler step
		it++;
		if (it < 40)
			verlet_int(vd,dt);
		else
		{
			euler_int(vd,dt);
			it = 0;
		}

		t += dt;

		if (write < t*10)
		{
			// Sensor pressure require update ghost, so we ensure that particles are distributed correctly
			// and ghost are updated
			vd.map(RUN_ON_DEVICE);
			vd.ghost_get<type,rho,Pressure,velocity>(RUN_ON_DEVICE);
			vd.updateCellList(NN);

			// calculate the pressure at the sensor points
			//sensor_pressure(vd,NN,press_t,probes);

			std::cout << "OUTPUT " << dt << std::endl;

			// When we write we have move all the particles information back to CPU

			vd.deviceToHostPos();
			vd.deviceToHostProp<type,rho,rho_prev,Pressure,drho,force,velocity,velocity_prev,red,red2>();

			// We copy on another vector with less properties to reduce the size of the output
			vector_dist_gpu<3,real_number,aggregate<unsigned int,real_number[3]>> vd_out(vd.getDecomposition(),0);

			auto ito = vd.getDomainIterator();

			while(ito.isNext())
			{
				auto p = ito.get();

				vd_out.add();

				vd_out.getLastPos()[0] = vd.getPos(p)[0];
				vd_out.getLastPos()[1] = vd.getPos(p)[1];
				vd_out.getLastPos()[2] = vd.getPos(p)[2];

				vd_out.template getLastProp<0>() = vd.template getProp<type>(p);

				vd_out.template getLastProp<1>()[0] = vd.template getProp<velocity>(p)[0];
				vd_out.template getLastProp<1>()[1] = vd.template getProp<velocity>(p)[1];
				vd_out.template getLastProp<1>()[2] = vd.template getProp<velocity>(p)[2];

				++ito;
			}

			vd_out.write_frame("Particles",write,VTK_WRITER | FORMAT_BINARY);
			write++;

			if (v_cl.getProcessUnitID() == 0)
			{std::cout << "TIME: " << t << "  write " << it_time.getwct() << "   " << it_reb << "   " << cnt << " Max visc: " << max_visc << "   " << vd.size_local()  << std::endl;}
		}
		else
		{
			if (v_cl.getProcessUnitID() == 0)
			{std::cout << "TIME: " << t << "  " << it_time.getwct() << "   " << it_reb << "   " << cnt  << " Max visc: " << max_visc << "   " << vd.size_local() << std::endl;}
		}
	}

	tot_sim.stop();
	std::cout << "Time to complete: " << tot_sim.getwct() << " seconds" << std::endl;


	openfpm_finalize();
}
 
#else

int main(int argc, char* argv[])
{
        return 0;
}

#endif
